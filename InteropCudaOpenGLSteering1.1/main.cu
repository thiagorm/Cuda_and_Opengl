#include "hip/hip_runtime.h"
#include <GLTools.h>
#include <stdlib.h>
#include <conio.h>
#include <time.h>

#ifdef __APPLE__
#include <glut/glut.h>
#else
#define FREEGLUT_STATIC
#include <GL\glut.h>
#endif

#define GL_GLEXT_PROTOTYPES 
#include  "hip/hip_runtime.h" 
#include  "cuda_gl_interop.h" 
#include  "common/book.h" 
#include "common/gl_helper.h"
#include "common/GL/glext.h"

#include "Vector3D.cu"
#include "AgentProperty.cu"

#define QUADS_LENTH 12
//Mudar a quantidade de agentes na tela
#define QTDE_AGENT 500


AgentProperty agents[QTDE_AGENT];

//AgentProperty agent(Vector3D(0.0, 0.0, 0.0), Vector3D(0.0, 0.0, 0.0));
AgentProperty target(Vector3D(0.05, 0.05, 0.0), Vector3D(0.0, 0.0, 0.0));
//AgentProperty steering;

float verticeLenth = 0.05;

float vVertsTarget[QUADS_LENTH]; 
float vVertsAgent[QUADS_LENTH]; 

float *vertsAgent;

float xstep = 0.02;
float ystep = 0.02;
float stepSize = 0.025f;

float wh = 1, ww = 1;


GLuint idVBOAgent[1];
GLuint idVBOTarget;
hipGraphicsResource *resource;

float *devPtr;
size_t size;

GLint first[QTDE_AGENT];
GLint count[QTDE_AGENT];


AgentProperty *dev_vector;

__device__ float length(float x, float y, float z)
	{
		return sqrtf(x*x + y*y + z*z);
	}

__device__ float3 Vec3DNormalize(float3 v)
{
	float3 vec;

	float vector_length = length(v.x,v.y,v.z);

	vec.x = v.x / vector_length;
	vec.y = v.y / vector_length;
	vec.z = v.z / vector_length;

	return vec;
}

__device__ float3 calculate_steering(float3 target, float3 agentPosition)
{
	float3 desired_velocity;
	float3 sub;

	sub.x = target.x - agentPosition.x;
	sub.y = target.y - agentPosition.y;
	sub.z = target.z - agentPosition.z;

	desired_velocity = Vec3DNormalize(sub);
	desired_velocity.x *= MAX_SPEED;
	desired_velocity.y *= MAX_SPEED;
	desired_velocity.z *= MAX_SPEED;
	return (desired_velocity);
}

__device__ float3 truncVector(float3 vector3, float max)
	{
		if(length(vector3.x, vector3.y, vector3.z) > max)
		{
			vector3 = Vec3DNormalize(vector3);
			vector3.x *= max;
			vector3.y *= max;
			vector3.z *= max;
		}
			
		return vector3;
	}


__global__ void desenha( float *vVerts, AgentProperty *agentPosition) 
{

	int x = blockDim.x * blockIdx.x + threadIdx.x;

	int offset = x;

	float verticeLenth = 0.05;

	vVerts[offset*12 + 0] = agentPosition[offset].position.x;
	vVerts[offset*12 + 1] = agentPosition[offset].position.y;
	vVerts[offset*12 + 2] = agentPosition[offset].position.z;

	vVerts[offset*12 + 3] = (agentPosition[offset].position.x)-verticeLenth;
	vVerts[offset*12 + 4] = agentPosition[offset].position.y;
	vVerts[offset*12 + 5] = agentPosition[offset].position.z;

	vVerts[offset*12 + 6] = agentPosition[offset].position.x;
	vVerts[offset*12 + 7] = agentPosition[offset].position.y-verticeLenth;
	vVerts[offset*12 + 8] = agentPosition[offset].position.z;

	vVerts[offset*12 + 9] = (agentPosition[offset].position.x)-verticeLenth;
	vVerts[offset*12 + 10] = agentPosition[offset].position.y-verticeLenth;
	vVerts[offset*12 + 11] = agentPosition[offset].position.z;

}

__global__ void kernel( float *vVerts, float verticeLenth, float stepSize, Vector3D targetPosition) 
{

	int x = blockDim.x * blockIdx.x + threadIdx.x;

	int offset = x;

	float3 agentPosition;
	float3 tPosition;

	tPosition.x = targetPosition.x;
	tPosition.y = targetPosition.y;
	tPosition.z = targetPosition.z;

	agentPosition.x = -vVerts[offset*12 + 0];
	agentPosition.y = vVerts[offset*12 + 1];
	agentPosition.z = vVerts[offset*12 + 2];

	float3 velocity = calculate_steering(tPosition, agentPosition);
	float3 force = truncVector(velocity, MAX_FORCE);

	float positionX = -vVerts[offset*12 + 0] + force.x * stepSize; 
	float positionY = vVerts[offset*12 + 1] + force.y * stepSize;
	//float positionZ = vVerts[offset*12 + 2] + force.z * stepSize;

	vVerts[offset*12 + 0] = -positionX;
	vVerts[offset*12 + 1] = positionY;
	//vVerts[offset*12 + 2] = positionZ;

	vVerts[offset*12 + 3] = -(positionX-verticeLenth);
	vVerts[offset*12 + 4] = positionY;
	//vVerts[offset*12 + 5] = positionZ;

	vVerts[offset*12 + 6] = -positionX;
	vVerts[offset*12 + 7] = (positionY-verticeLenth);
	//vVerts[offset*12 + 8] = positionZ;

	vVerts[offset*12 + 9] = -(positionX-verticeLenth);
	vVerts[offset*12 + 10] = (positionY-verticeLenth);
	//vVerts[offset*12 + 11] = positionZ;
   
}

void OnReshape(int w, int h)
{
	glViewport(0,0,w,h);
}

void SetAgent(AgentProperty *agents)
{

	//Essa fun��o pega o endere�o atual da memoria do device onde est� o buffer
	HANDLE_ERROR(hipGraphicsMapResources(1, &resource, NULL));

	//depois que o endere�o eh pego, o mesmo � salvo no ponteiro devPtr
	HANDLE_ERROR(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, resource));


	int div = 2;
	int blocks = QTDE_AGENT/div;
	desenha<<<blocks, div>>>(devPtr, agents);

	HANDLE_ERROR( hipGraphicsUnmapResources( 1, &resource, NULL ) );
}

void DrawAgent()
{
	//Essa fun��o pega o endere�o atual da memoria do device onde est� o buffer
	HANDLE_ERROR(hipGraphicsMapResources(1, &resource, NULL));

	//depois que o endere�o eh pego, o mesmo � salvo no ponteiro devPtr
	HANDLE_ERROR(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, resource));


	int div = 2;
	int blocks = QTDE_AGENT/div;
	kernel<<<blocks, div>>>(devPtr, verticeLenth, stepSize, target.position);

	HANDLE_ERROR( hipGraphicsUnmapResources( 1, &resource, NULL ) );

	glBindBuffer(GL_ARRAY_BUFFER, idVBOAgent[0]);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, 0);

	int size = QTDE_AGENT;
	glEnableClientState(GL_VERTEX_ARRAY);
	glMultiDrawArrays(GL_TRIANGLE_STRIP, first, count, size);
	glDisableClientState(GL_VERTEX_ARRAY);
}

void Init()
{

	glClear(GL_COLOR_BUFFER_BIT);
	glClearColor(0.0f, 0.5f, 1.0f, 1.0f);

	target.AgentMovement(vVertsTarget, target.position, verticeLenth);

	glGenBuffers(QTDE_AGENT, idVBOAgent);
	glGenBuffers(1, &idVBOTarget);

	glBindBuffer(GL_ARRAY_BUFFER, idVBOTarget);
	glBufferData(GL_ARRAY_BUFFER, QUADS_LENTH*sizeof(float), vVertsTarget, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, idVBOAgent[0]);
	int size = (QUADS_LENTH*QTDE_AGENT) * sizeof(float);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	//Essa fun��o serve para atribuir o bufferObj do opengl ao buffer resource do CUDA
	//a partir dessa fun��o o CUDA pode trabalhar com o buffer do opengl
	HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&resource, idVBOAgent[0], cudaGraphicsMapFlagsWriteDiscard));

	float r1, r2;
	int r;

	hipMalloc( (void**)&dev_vector, QTDE_AGENT *  sizeof(AgentProperty ) );

	for(int i = 0; i < QTDE_AGENT; i++)
	{
		r = rand()%4 + 1;
		switch(r)
		{
		case 1:
			r1 = (float) rand()/(float)RAND_MAX;
			r2 = (float) rand()/(float)RAND_MAX;
			break;
		case 2:
			r1 = -(float) rand()/(float)RAND_MAX;
			r2 = (float) rand()/(float)RAND_MAX;
			break;
		case 3:
			r1 = (float) rand()/(float)RAND_MAX;
			r2 = -(float) rand()/(float)RAND_MAX;
			break;
		case 4:
			r1 = -(float) rand()/(float)RAND_MAX;
			r2 = -(float) rand()/(float)RAND_MAX;
			break;
		}
		agents[i] = AgentProperty(Vector3D(r1,r2,0.0), Vector3D(0.0,0.0,0.0));
	}

	/*for(int i = 0; i < QTDE_AGENT; i++)
	{
		printf("%d\n", i);
		printf("x: %.2f\n", agents[i].position.x);
		printf("y: %.2f\n", agents[i].position.y);
		printf("z: %.2f\n", agents[i].position.z);
		printf("\n");
	}*/

	hipMemcpy( dev_vector, agents, QTDE_AGENT * sizeof(AgentProperty ), hipMemcpyHostToDevice );

	SetAgent(dev_vector);

	for(int j = 0; j < QTDE_AGENT; j++)
	{
		if(j == 0)
			first[j] = 0;
		else
		{
			first[j] = 4*j;
		}

		count[j] = 4;
	}

}

void Render()
{
	glClear(GL_COLOR_BUFFER_BIT);

	glEnableVertexAttribArray(0);

	glBindBuffer(GL_ARRAY_BUFFER, idVBOTarget);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, 0);
	glDrawArrays(GL_TRIANGLE_STRIP, 0, 4);

	DrawAgent();

	glutSwapBuffers();

	glutPostRedisplay();
}


static void OnKeyboard(int key, int x, int y)
{

	if(key == GLUT_KEY_UP)
	{
		target.position.y += stepSize;
	}

	if(key == GLUT_KEY_DOWN)
	{
		target.position.y -= stepSize;
	}

	if(key == GLUT_KEY_LEFT)
	{
		target.position.x += stepSize;
	}

	if(key == GLUT_KEY_RIGHT)
	{
		target.position.x -= stepSize;
	}

	if(key == GLUT_KEY_END)
	{
		HANDLE_ERROR( hipGraphicsUnregisterResource( resource ) );

		glBindBuffer( GL_ARRAY_BUFFER, 0 );
		glDeleteBuffers(QTDE_AGENT, idVBOAgent);

		glBindBuffer( GL_ARRAY_BUFFER, 0 );
		glDeleteBuffers(1, &idVBOTarget);

		hipFree(dev_vector);

		exit(1);
	}

	target.AgentMovement(vVertsTarget, target.position, verticeLenth);

	glBindBuffer(GL_ARRAY_BUFFER, idVBOTarget);
	glBufferData(GL_ARRAY_BUFFER, QUADS_LENTH*sizeof(float), vVertsTarget, GL_DYNAMIC_DRAW);

	glutPostRedisplay(); 
}

int main(int argc, char *argv[])
{

	srand ( time(NULL) );

	//M�todo para escolher uma das GPU's se o sistema tiver mais de uma, que tiver 
	//um poder computacional de 1.0 ou melhor
	hipDeviceProp_t  deviceProp;
	int dev = NULL;

	memset(&deviceProp, 0, sizeof(hipDeviceProp_t));
	deviceProp.major = 1;
	deviceProp.minor = 0;

	HANDLE_ERROR(hipChooseDevice(&dev, &deviceProp));

	//Indicar qual � o device(Gpu) que vai ser utilizado para o trabalho de interoperabilidade
	//Essa fun��o � obrigatoria, quando se for usar a interoperabilidade entre OPENGL e CUDA
	HANDLE_ERROR(cudaGLSetGLDevice(dev));

	//Inicializando o Opengl
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(800, 600);
	glutCreateWindow("Square");

	//Loop
	glutReshapeFunc(OnReshape);
	glutDisplayFunc(Render);
	glutSpecialFunc(OnKeyboard);
	//

	GLenum err = glewInit();
	if(GLEW_OK != err)
	{
		fprintf(stderr, "Glew error: %s\n", glewGetErrorString(err));
		return 1;
	}

	//Init
	Init();

	//StartLoop
	glutMainLoop();

	return 0;
}